#include "hip/hip_runtime.h"
// -*- c++ -*-
/*
* find_neighbors.cu
*
*  
*
*
*/

#include <hip/hip_runtime.h>
#include "spherocyl_box.h"
#include "cudaErr.h"
#include "data_primitives.h"
#include <math.h>

using namespace std;

const double D_PI = 3.14159265358979;


__global__ void find_moi(int nSpherocyls, double *pdMOI, double *pdA)
{
  int thid = threadIdx.x + blockIdx.x * blockDim.x;

  while (thid < nSpherocyls) {
    double dA = pdA[thid];

    pdMOI[thid] = dA*dA / 3;
    
    thid += blockDim.x * gridDim.x;
  }
}

__global__ void find_rot_consts(int nSpherocyls, double *pdMOI, double *pdIsoCoeff, double *pdR, double *pdA)
{
  int thid = threadIdx.x + blockIdx.x * blockDim.x;

  while (thid < nSpherocyls) {
    double dR = pdR[thid];
    double dA = pdA[thid];
    double dAlpha = dA/dR;
    
    double dC = 3*D_PI + 24*dAlpha + 6*D_PI*dAlpha*dAlpha + 8*dAlpha*dAlpha*dAlpha;
    double dB = D_PI*dR*dR + 4*dR*dA;
    pdMOI[thid] = dR*dR*dR*dR*dC/(6*dB);
    pdIsoCoeff[thid] = (8*dAlpha + 6*D_PI*dAlpha*dAlpha + 8*dAlpha*dAlpha*dAlpha)/dC;

    thid += blockDim.x*gridDim.x;
  }
}



///////////////////////////////////////////////////////////////
// Find the Cell ID for each particle:
//  The list of cell IDs for each particle is returned to pnCellID
//  A list of which particles are in each cell is returned to pnCellList
//
// *NOTE* if there are more than nMaxPPC particles in a given cell,
//  not all of these particles will get added to the cell list
///////////////////////////////////////////////////////////////
__global__ void find_cells(int nSpherocyls, int nMaxPPC, double dCellW, double dCellH,
			   int nCellCols, double dL, double *pdX, double *pdY, 
			   int *pnCellID, int *pnPPC, int *pnCellList)
{
  // Assign each thread a unique ID accross all thread-blocks, this is its particle ID
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nSpherocyls) {
    double dX = pdX[nPID];
    double dY = pdY[nPID];
    
    // I often allow the stored coordinates to drift slightly outside the box limits
    //  until 
    if (dY > dL)
      {
	dY -= dL;
	pdY[nPID] = dY;
      }
    else if (dY < 0)
      {
	dY += dL;
	pdY[nPID] = dY;
      }
    if (dX > dL)
      {
	dX -= dL;
	pdX[nPID] = dX;
      }
    else if (dX < 0)
      {
	dX += dL;
	pdX[nPID] = dX;
      }

    //find the cell ID, add a particle to that cell 
    int nCol = (int)(dX / dCellW);
    int nRow = (int)(dY / dCellH); 
    int nCellID = nCol + nRow * nCellCols;
    pnCellID[nPID] = nCellID;

    // Add 1 particle to a cell safely (only allows one thread to access the memory
    //  address at a time). nPPC is the original value, not the result of addition 
    int nPPC = atomicAdd(pnPPC + nCellID, 1);
    
    // only add particle to cell if there is not already the maximum number in cell
    if (nPPC < nMaxPPC)
      pnCellList[nCellID * nMaxPPC + nPPC] = nPID;
    else
      nPPC = atomicAdd(pnPPC + nCellID, -1);

    nPID += nThreads;
  }
}


////////////////////////////////////////////////////////////////
// Here a list of possible contacts is created for each particle
//  The list of neighbors is returned to pnNbrList
//
// This is one function that I may target for optimization in
//  the future because I know it is slowed down by "branch divergence"
////////////////////////////////////////////////////////////////
__global__ void find_nbrs(int nSpherocyls, int nMaxPPC, int *pnCellID, int *pnPPC, 
			  int *pnCellList, int *pnAdjCells, int nMaxNbrs, int *pnNPP, 
			  int *pnNbrList, double *pdX, double *pdY, double *pdR, 
			  double *pdA, double dEpsilon, double dL, double dGamma)
{
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = gridDim.x * blockDim.x;

  while (nPID < nSpherocyls)
    {
      double dX = pdX[nPID];
      double dY = pdY[nPID];
      double dR = pdR[nPID];
      double dA = pdA[nPID];
      int nNbrs = 0;

      // Particles in adjacent cells are added if they are close enough to 
      //  interact without each moving by more than dEpsilon/2
      int nCellID = pnCellID[nPID];
      int nP = pnPPC[nCellID];
      for (int p = 0; p < nP; p++)
	{
	  int nAdjPID = pnCellList[nCellID*nMaxPPC + p];
	  if (nAdjPID != nPID)
	    {
	      double dSigma = dR + dA + pdR[nAdjPID] + pdA[nAdjPID] + dEpsilon;
	      double dDeltaY = dY - pdY[nAdjPID];
	      dDeltaY += dL * ((dDeltaY < -0.5 * dL) - (dDeltaY > 0.5 * dL));
	      
	      if (fabs(dDeltaY) < dSigma)
		{
		  double dDeltaX = dX - pdX[nAdjPID];
		  dDeltaX += dL * ((dDeltaX < -0.5 * dL) - (dDeltaX > 0.5 * dL));
		  double dDeltaRx = dDeltaX + dGamma * dDeltaY;
		  double dDeltaRx2 = dDeltaX + 0.5 * dDeltaY;
		  if (fabs(dDeltaRx) < dSigma || fabs(dDeltaRx2) < dSigma)
		    {
		      // This indexing makes global memory accesses more coalesced
		      if (nNbrs < nMaxNbrs)
			{
			  pnNbrList[nSpherocyls * nNbrs + nPID] = nAdjPID;
			  nNbrs += 1;
			}
		    }
		}
	    }
	}

      for (int nc = 0; nc < 8; nc++)
	{
	  int nAdjCID = pnAdjCells[8 * nCellID + nc];
	  nP = pnPPC[nAdjCID];
	  for (int p = 0; p < nP; p++)
	    {
	      int nAdjPID = pnCellList[nAdjCID*nMaxPPC + p];
	      // The maximum distance at which two particles could contact
	      //  plus a little bit of moving room - dEpsilon 
	      double dSigma = dR + dA + pdA[nAdjPID] + pdR[nAdjPID] + dEpsilon;
	      double dDeltaY = dY - pdY[nAdjPID];
		
	      // Make sure were finding the closest separation
	      dDeltaY += dL * ((dDeltaY < -0.5 * dL) - (dDeltaY > 0.5 * dL));
	      
	      if (fabs(dDeltaY) < dSigma)
		{
		  double dDeltaX = dX - pdX[nAdjPID];
		  dDeltaX += dL * ((dDeltaX < -0.5 * dL) - (dDeltaX > 0.5 * dL));
		  
		  // Go to unsheared coordinates
		  double dDeltaRx = dDeltaX + dGamma * dDeltaY;
		  // Also look at distance when the strain parameter is at its max (0.5)
		  double dDeltaRx2 = dDeltaX + 0.5 * dDeltaY;
		  if (fabs(dDeltaRx) < dSigma || fabs(dDeltaRx2) < dSigma)
		    {
		      if (nNbrs < nMaxNbrs)
			{
			  pnNbrList[nSpherocyls * nNbrs + nPID] = nAdjPID;
			  nNbrs += 1;
			}
		    }
		}
	    }   
	}
      
      pnNPP[nPID] = nNbrs;
      nPID += nThreads;
    }
}



///////////////////////////////////////////////////////////////
// Finds a list of possible contacts for each particle
//
// Usually when things are moving I keep track of an Xmoved and Ymoved
//  and only call this to make a new list of neighbors if some particle
//  has moved more than (dEpsilon / 2) in some direction
///////////////////////////////////////////////////////////////
void Spherocyl_Box::find_neighbors()
{
  // reset each byte to 0
  hipMemset((void *) d_pnPPC, 0, sizeof(int)*m_nCells);
  hipMemset((void *) d_pdXMoved, 0, sizeof(double)*m_nSpherocyls);
  hipMemset((void *) d_pdYMoved, 0, sizeof(double)*m_nSpherocyls);
  hipMemset((void *) d_bNewNbrs, 0, sizeof(int));

  if (!m_bMOI)
    find_rot_consts <<<m_nGridSize, m_nBlockSize>>> (m_nSpherocyls, d_pdMOI, d_pdIsoC, d_pdR, d_pdA);

  find_cells <<<m_nGridSize, m_nBlockSize>>>
    (m_nSpherocyls, m_nMaxPPC, m_dCellW, m_dCellH, m_nCellCols, 
     m_dL, d_pdX, d_pdY, d_pnCellID, d_pnPPC, d_pnCellList);
  hipDeviceSynchronize();
  checkCudaError("Finding cells");


  find_nbrs <<<m_nGridSize, m_nBlockSize>>>
    (m_nSpherocyls, m_nMaxPPC, d_pnCellID, d_pnPPC, d_pnCellList, d_pnAdjCells,
     m_nMaxNbrs, d_pnNPP, d_pnNbrList, d_pdX, d_pdY, d_pdR, d_pdA, 
     m_dEpsilon, m_dL, m_dGamma);
  hipDeviceSynchronize();
  checkCudaError("Finding neighbors");

  /*
  int *h_pnCellID = (int*) malloc(sizeof(int)*3*m_nSpherocyls);
  int *h_pnNPP = (int*) malloc(sizeof(int)*3*m_nSpherocyls);
  int *h_pnNbrList = (int*) malloc(sizeof(int)*3*m_nSpherocyls*m_nMaxNbrs);
  hipMemcpy(h_pnCellID, d_pnCellID, sizeof(int)*3*m_nSpherocyls, hipMemcpyDeviceToHost);
  hipMemcpy(h_pnNPP,d_pnNPP, sizeof(int)*3*m_nSpherocyls,hipMemcpyDeviceToHost);
  hipMemcpy(h_pnNbrList, d_pnNbrList, sizeof(int)*3*m_nSpherocyls*m_nMaxNbrs, hipMemcpyDeviceToHost);

  for (int p = 0; p < 3*m_nSpherocyls; p++) {
    printf("Spherocyl: %d, Cell: %d, neighbors: %d\n", 
	   p, h_pnCellID[p], h_pnNPP[p]);
    for (int n = 0; n < h_pnNPP[p]; n++) {
      printf("%d ", h_pnNbrList[n*3*m_nSpherocyls + p]);
    }
    printf("\n");
    fflush(stdout);
  }

  free(h_pnCellID); free(h_pnNPP); free(h_pnNbrList);
  */
}


////////////////////////////////////////////////////////////////////////////////////
// Sets gamma back by 1 (used when gamma > 0.5)
//  also finds the cells in the process
//
///////////////////////////////////////////////////////////////////////////////////
__global__ void set_back_coords(int nSpherocyls, double dL, double *pdX, double *pdY, double *pdPhi)
{
  // Assign each thread a unique ID accross all thread-blocks, this is its particle ID
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nSpherocyls) {
    double dX = pdX[nPID];
    double dY = pdY[nPID];
    double dPhi = pdPhi[nPID];
    
    // I often allow the stored coordinates to drift slightly outside the box limits
    if (dPhi > D_PI)
      {
	dPhi -= 2*D_PI;
	pdPhi[nPID] = dPhi;
      }
    else if (dPhi < -D_PI)
      {
	dPhi += 2*D_PI;
	pdPhi[nPID] = dPhi;
      }
    if (dY > dL)
      {
	dY -= dL;
	pdY[nPID] = dY;
      }
    else if (dY < 0)
      {
	dY += dL;
	pdY[nPID] = dY;
      }
    
    // When gamma -> gamma-1, Xi -> Xi + Yi
    dX += dY;
    if (dX < 0)
      {
	dX += dL;
      }
    while (dX > dL)
      {
	dX -= dL;
      }
    pdX[nPID] = dX;

    nPID += nThreads;
  }
  

}


void Spherocyl_Box::set_back_gamma()
{
  hipMemset((void *) d_pnPPC, 0, sizeof(int)*m_nCells);
  hipMemset((void *) d_pdXMoved, 0, sizeof(double)*m_nSpherocyls);
  hipMemset((void *) d_pdYMoved, 0, sizeof(double)*m_nSpherocyls);
  hipMemset((void *) d_bNewNbrs, 0, sizeof(int));

  /*
  int *h_pnCellID = (int*) malloc(sizeof(int)*m_nSpherocyls);
  int *h_pnNPP = (int*) malloc(sizeof(int)*m_nSpherocyls);
  int *h_pnNbrList = (int*) malloc(sizeof(int)*m_nSpherocyls*m_nMaxNbrs);
  hipMemcpy(h_pnCellID, d_pnCellID, sizeof(int)*m_nSpherocyls, hipMemcpyDeviceToHost);
  hipMemcpy(h_pnNPP,d_pnNPP, sizeof(int)*m_nSpherocyls,hipMemcpyDeviceToHost);
  hipMemcpy(h_pnNbrList, d_pnNbrList, sizeof(int)*m_nSpherocyls*m_nMaxNbrs, hipMemcpyDeviceToHost);

  printf("\nSetting coordinate system back by gamma\n\nOld neighbors:");
  for (int p = 0; p < m_nSpherocyls; p++) {
    printf("Spherocyl: %d, Cell: %d, neighbors: %d\n", 
	   p, h_pnCellID[p], h_pnNPP[p]);
    for (int n = 0; n < h_pnNPP[p]; n++) {
      printf("%d ", h_pnNbrList[n*m_nSpherocyls + p]);
    }
    printf("\n");
    fflush(stdout);
  }
*/

  set_back_coords <<<m_nGridSize, m_nBlockSize>>> 
    (m_nSpherocyls, m_dL, d_pdX, d_pdY, d_pdPhi);
  hipDeviceSynchronize();
  checkCudaError("Finding new coordinates, cells");
  m_dGamma -= 1;
  m_dTotalGamma = int(m_dTotalGamma+1) + m_dGamma;  // Gamma total will have diverged slightly due to differences in precision with gamma

  find_neighbors();

  /*
  hipMemcpy(h_pnCellID, d_pnCellID, sizeof(int)*m_nSpherocyls, hipMemcpyDeviceToHost);
  hipMemcpy(h_pnNPP,d_pnNPP, sizeof(int)*m_nSpherocyls,hipMemcpyDeviceToHost);
  hipMemcpy(h_pnNbrList, d_pnNbrList, sizeof(int)*m_nSpherocyls*m_nMaxNbrs, hipMemcpyDeviceToHost);
  printf("\nNew Neighbors:\n");
  for (int p = 0; p < m_nSpherocyls; p++) {
    printf("Spherocyl: %d, Cell: %d, neighbors: %d\n", 
	   p, h_pnCellID[p], h_pnNPP[p]);
    for (int n = 0; n < h_pnNPP[p]; n++) {
      printf("%d ", h_pnNbrList[n*m_nSpherocyls + p]);
    }
    printf("\n");
    fflush(stdout);
  }
  
  free(h_pnCellID); free(h_pnNPP); free(h_pnNbrList);
  */
}


////////////////////////////////////////////////////////////////////////////
// Finds cells for all particles regardless of maximum particle per cell
//  used for reordering particles
/////////////////////////////////////////////////////////////////////////
__global__ void find_cells_nomax(int nSpherocyls, double dCellW, double dCellH,
				 int nCellCols, double dL, double *pdX, double *pdY, 
				 int *pnCellID, int *pnPPC)
{
  // Assign each thread a unique ID accross all thread-blocks, this is its particle ID
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nSpherocyls) {
    double dX = pdX[nPID];
    double dY = pdY[nPID];
    
    // Particles are allowed to drift slightly outside the box limits
    //  until cells are reassigned due to a particle drift of dEpsilon/2 
    if (dY > dL) {
      dY -= dL; 
      pdY[nPID] = dY; }
    else if (dY < 0) {
      dY += dL;
      pdY[nPID] = dY; }
    if (dX > dL) {
      dX -= dL; 
      pdX[nPID] = dX; }
    else if (dX < 0) {
      dX += dL;
      pdX[nPID] = dX; }

    //find the cell ID, add a particle to that cell 
    int nCol = (int)(dX / dCellW);
    int nRow = (int)(dY / dCellH); 
    int nCellID = nCol + nRow * nCellCols;
    
    pnCellID[nPID] = nCellID;
    int nPPC = atomicAdd(pnPPC + nCellID, 1);
    
    nPID += nThreads; }
}

__global__ void reorder_part(int nSpherocyls, double *pdTempX, double *pdTempY,
			     double *pdTempR, double *pdTempA, 
			     int *pnTempInitID, double *pdX, double *pdY, 
			     double *pdR, double *pdA, int *pnInitID, 
			     int *pnMemID, int *pnCellID, int *pnCellSID)
{
  int nPID = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (nPID < nSpherocyls) {
    double dX = pdTempX[nPID];
    double dY = pdTempY[nPID];
    double dR = pdTempR[nPID];
    double dA = pdTempA[nPID];
    int nInitID = pnTempInitID[nPID];

    int nCellID = pnCellID[nPID];
    int nNewID = atomicAdd(pnCellSID + nCellID, 1);
    
    pdX[nNewID] = dX;
    pdY[nNewID] = dY;
    pdR[nNewID] = dR;
    pdA[nNewID] = dA;
    pnMemID[nInitID] = nNewID;
    pnInitID[nNewID] = nInitID;

    nPID += nThreads; 
  }
}

__global__ void invert_IDs(int nIDs, int *pnIn, int *pnOut)
{
  int thid = threadIdx.x + blockIdx.x * blockDim.x;
  int nThreads = blockDim.x * gridDim.x;

  while (thid < nIDs) {
    int i = pnIn[thid];
    pnOut[i] = thid; 
    thid += nThreads; }
    
}

void Spherocyl_Box::reorder_particles()
{
  hipMemset((void *) d_pnPPC, 0, sizeof(int)*m_nCells);

  //find particle cell IDs and number of particles in each cell
  find_cells_nomax <<<m_nGridSize, m_nBlockSize>>>
    (m_nSpherocyls, m_dCellW, m_dCellH, m_nCellCols, 
     m_dL, d_pdX, d_pdY, d_pnCellID, d_pnPPC);
  hipDeviceSynchronize();
  checkCudaError("Reordering particles: Finding cells");

  int *d_pnCellSID;
  int *d_pnTempInitID;
  double *d_pdTempR; 
  double *d_pdTempA; 
  hipMalloc((void **) &d_pnCellSID, sizeof(int) * m_nCells);
  hipMalloc((void **) &d_pdTempR, sizeof(double) * m_nSpherocyls);
  hipMalloc((void **) &d_pdTempA, sizeof(double) * m_nSpherocyls);
  hipMalloc((void **) &d_pnTempInitID, sizeof(int) * m_nSpherocyls);
  hipMemcpy(d_pdTempX, d_pdX, sizeof(double) * m_nSpherocyls, hipMemcpyDeviceToDevice);
  hipMemcpy(d_pdTempY, d_pdY, sizeof(double) * m_nSpherocyls, hipMemcpyDeviceToDevice);
  hipMemcpy(d_pdTempR, d_pdR, sizeof(double) * m_nSpherocyls, hipMemcpyDeviceToDevice);
  hipMemcpy(d_pdTempA, d_pdA, sizeof(double) * m_nSpherocyls, hipMemcpyDeviceToDevice);
  hipMemcpy(d_pnTempInitID, d_pnInitID, sizeof(int) * m_nSpherocyls, hipMemcpyDeviceToDevice);

  exclusive_scan(d_pnPPC, d_pnCellSID, m_nCells);

  /*
  int *h_pnCellSID = (int*) malloc(m_nCells * sizeof(int));
  int *h_pnCellNPart = (int*) malloc(m_nCells * sizeof(int));
  hipMemcpy(h_pnCellNPart, d_pnCellNPart, sizeof(int)*m_nCells, hipMemcpyDeviceToHost);
  hipMemcpy(h_pnCellSID, d_pnCellSID, sizeof(int)*m_nCells, hipMemcpyDeviceToHost);
  for (int c = 0; c < m_nCells; c++)
    {
      printf("%d %d\n", h_pnCellNPart[c], h_pnCellSID[c]);
    }
  free(h_pnCellSID);
  free(h_pnCellNPart);
  */

  //reorder particles based on cell ID (first by Y direction)
  reorder_part <<<m_nGridSize, m_nBlockSize>>>
    (m_nSpherocyls, d_pdTempX, d_pdTempY, d_pdTempR, d_pdTempA, d_pnTempInitID,
     d_pdX, d_pdY, d_pdR, d_pdA, d_pnInitID, d_pnMemID, d_pnCellID, d_pnCellSID);
  hipDeviceSynchronize();
  checkCudaError("Reordering particles: changing order");

  //invert_IDs <<<m_nGridSize, m_nBlockSize>>> (m_nSpherocyls, d_pnMemID, d_pnInitID);
  hipMemcpyAsync(h_pnMemID, d_pnMemID, m_nSpherocyls*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpyAsync(h_pdR, d_pdR, m_nSpherocyls*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpyAsync(h_pdA, d_pdA, m_nSpherocyls*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  hipFree(d_pnCellSID); hipFree(d_pnTempInitID);
  hipFree(d_pdTempR); hipFree(d_pdTempA);

  m_bMOI = 0;
  find_neighbors();
}


////////////////////////////////////////////////////////////////////////
// Sets the particle IDs to their order in memory
//  so the current IDs become the initial IDs
/////////////////////////////////////////////////////////////////////
void Spherocyl_Box::reset_IDs()
{
  ordered_array(d_pnInitID, m_nSpherocyls, m_nGridSize, m_nBlockSize);
  hipMemcpyAsync(h_pnMemID, d_pnInitID, sizeof(int)*m_nSpherocyls, hipMemcpyDeviceToHost);
  hipMemcpy(d_pnMemID, d_pnInitID, sizeof(int)*m_nSpherocyls, hipMemcpyDeviceToDevice);
  
}
